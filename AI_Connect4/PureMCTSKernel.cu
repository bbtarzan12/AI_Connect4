#include "hip/hip_runtime.h"
#include "PureMCTSKernel.cuh"

#include "Player.h"
#include "Map.h"
#include "Random.h"

#include <iostream>
#include "CommandLineRenderer.h"

using namespace std;

#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define PSEUDO_FLOOR( V ) ((V) >= 0 ? (int)(V) : (int)((V) - 1))

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__device__ uint32_t xor128()
{
	static uint32_t seed = 1;
	uint32_t t = 3906;
	seed ^= t << 15;
	seed ^= seed << 13;
	seed ^= seed >> 17;
	seed ^= seed << 5;
	return seed;
}

__device__ int Random(int min, int max)
{
	return PSEUDO_FLOOR(((double)xor128() / 0xffffffff) * (max - min + 1)) + min;
}

__host__ __device__ inline int Convert2Dto1D(const Column column, const Row row)
{
	return row * MAX_COLUMN + column;
}

__device__ GPUCoord GetEmptyCoord(ID* map, const Column column, bool& isValid)
{
	for (Row row = 0; row < MAX_ROW; row++)
	{
		if (map[Convert2Dto1D(column, row)] != EMPTY_ID)
			continue;

		isValid = true;
		return GPUCoord(column, row);
	}

	isValid = false;
	return GPUCoord();
}

__device__ inline bool IsColumnValid(ID* map, const Column column)
{
	return map[Convert2Dto1D(column, MAX_ROW - 1)] == EMPTY_ID;
}

__device__ bool IsMapFull(ID* map)
{
	for (Column column = 0; column < MAX_COLUMN; column++)
	{
		if (map[Convert2Dto1D(column, MAX_ROW - 1)] == EMPTY_ID)
			return false;
	}
	return true;
}

__device__ inline bool CheckCoordIsInBound(const Column column, const Row row)
{
	//if (column >= MAX_COLUMN || column < 0)
	//	return false;

	//if (row >= MAX_ROW || row < 0)
	//	return false;

	//return true;

	return !((column >= MAX_COLUMN || column < 0) || ((row >= MAX_ROW || row < 0)));
}

__device__ int GetNumOfNeighbor(ID* map, const GPUCoord originCoord, const int deltaColumn, const int deltaRow, const ID id)
{
	GPUCoord neighborCoord;
	neighborCoord.column = originCoord.column + deltaColumn;
	neighborCoord.row = originCoord.row + deltaRow;

	if (!CheckCoordIsInBound(neighborCoord.column, neighborCoord.row))
		return 0;

	if (map[Convert2Dto1D(neighborCoord.column, neighborCoord.row)] == id)
		return GetNumOfNeighbor(map, neighborCoord, deltaColumn, deltaRow, id) + 1;

	return 0;
}

__device__ void GetNumOfNeighbors(ID* map, const GPUCoord coord, const ID id, int* neighbors, bool selfContained = true)
{
	const Column& column = coord.column;
	const Row& row = coord.row;

	int index = 0;
	for (int neighborColumn = column - 1; neighborColumn <= column + 1; neighborColumn++)
	{
		for (int neighborRow = row - 1; neighborRow <= row; neighborRow++)
		{
			if (neighborColumn == column && neighborRow == row)
				continue;

			if (neighborColumn == column + 1 && neighborRow == row)
				continue;

			int deltaColumn = neighborColumn - column;
			int deltaRow = neighborRow - row;
			int numOfNeighbor = GetNumOfNeighbor(map, coord, deltaColumn, deltaRow, id) + GetNumOfNeighbor(map, coord, -deltaColumn, -deltaRow, id) + (selfContained ? 1 : 0);
			neighbors[index++] = numOfNeighbor;
		}
	}
}
 
__device__ bool IsGameEnd(ID* map, const GPUCoord coord, const ID id)
{
	int numOfNeighbors[4] = { 0, 0, 0, 0 };
	GetNumOfNeighbors(map, coord, id, &numOfNeighbors[0]);

	int maxNeighbor = -1;
	for (int i = 0; i < 4; i++)
	{
		if(maxNeighbor > numOfNeighbors[i])
			continue;

		maxNeighbor = numOfNeighbors[i];

		if (maxNeighbor >= 4)
			return true;
	}
	return false;
}

__device__ void SetCoord(ID* map, const ID id, const GPUCoord coord, GPUCoord& lastMove)
{
	if (!CheckCoordIsInBound(coord.column, coord.row))
		return;
	lastMove = coord;
	map[Convert2Dto1D(coord.column, coord.row)] = id;
}

__global__ void SimulationKernel(ID* globalMap, const KernelData kernelData, int* numSimulation, int* totalWin)
{
	ID currentID = kernelData.id;
	ID winID = currentID;
	GPUCoord lastMove = kernelData.lastMove;

	ID map[MAX_COLUMN * MAX_ROW];

	for (Column column = 0; column < MAX_COLUMN; column++)
	{
		for (Row row = 0; row < MAX_ROW; row++)
		{
			int index = Convert2Dto1D(column, row);
			map[index] = globalMap[index];
		}
	}

	while (true)
	{
		if (IsGameEnd(map, lastMove, currentID))
		{
			winID = currentID;
			break;
		}

		if (IsMapFull(map))
		{
			winID = EMPTY_ID;
			break;
		}

		currentID = currentID ^ 1;
		Column validColumns[MAX_COLUMN];
		int numOfValidColumns = 0;

		for (Column column = 0; column < MAX_COLUMN; column++)
		{
			if (!IsColumnValid(map, column))
				continue;

			validColumns[numOfValidColumns++] = column;
		}

		if (numOfValidColumns <= 0)
		{
			printf("Un Error Occured 1 %d\n", numOfValidColumns);
			return;
		}

		int randomIndex = Random(0, numOfValidColumns - 1);
		if (randomIndex >= numOfValidColumns)
		{
			printf("randomIndex >= numOfValidColumns\n");
			return;
		}

		Column randomColumn = validColumns[randomIndex];

		bool isValid = false;
		GPUCoord randomCoord = GetEmptyCoord(map, randomColumn, isValid);
		if (!isValid)
		{
			printf("Un Error Occured 2 randomColumn : %d \n", randomColumn);
			return;
		}

		SetCoord(map, currentID, randomCoord, lastMove);
	}

	if (winID == kernelData.id)
	{
		atomicAdd(totalWin, 1);
	}
	atomicAdd(numSimulation, 1);
}

void Simulation(Map& childMap, const ID id, int maxSimulation, int* numSimulation, int* totalWin)
{
	const int block = 1024;
	const int grid = ceil((float)maxSimulation / block);

	ID cpuMap[MAX_COLUMN * MAX_ROW];
	
	for (Column column = 0; column < MAX_COLUMN; column++)
	{
		for (Row row = 0; row < MAX_ROW; row++)
		{
			cpuMap[Convert2Dto1D(column, row)] = childMap[column][row];
		}
	}

	int *gpuNumSimuation, *gpuTotalWin;
	gpuErrorCheck(hipMalloc(&gpuNumSimuation, sizeof(int)));
	gpuErrorCheck(hipMalloc(&gpuTotalWin, sizeof(int)));
	gpuErrorCheck(hipMemcpy(gpuNumSimuation, numSimulation, sizeof(int), hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(gpuTotalWin, totalWin, sizeof(int), hipMemcpyHostToDevice));

	ID *gpuMap;
	gpuErrorCheck(hipMalloc(&gpuMap, sizeof(ID) * MAX_COLUMN * MAX_ROW));
	gpuErrorCheck(hipMemcpy(gpuMap, cpuMap, sizeof(ID) * MAX_COLUMN * MAX_ROW, hipMemcpyHostToDevice));

	KernelData kernelData;
	kernelData.id = id;
	kernelData.lastMove.column = childMap.lastMove.first;
	kernelData.lastMove.row = childMap.lastMove.second;

	SimulationKernel << <grid, block >> > (gpuMap, kernelData, gpuNumSimuation, gpuTotalWin);
	gpuErrorCheck(hipDeviceSynchronize());

	gpuErrorCheck(hipMemcpy(numSimulation, gpuNumSimuation, sizeof(int), hipMemcpyDeviceToHost));
	gpuErrorCheck(hipMemcpy(totalWin, gpuTotalWin, sizeof(int), hipMemcpyDeviceToHost));
}
